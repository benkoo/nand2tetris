/**
 * Matrix multiplication Exercise : P = M . N.
 *
 * This program basically follows the tutorial in class.
 *
 * Given the 1024*1024 matrix test case, this program got the best performance
 * boost using TILE_WIDTH = 16. This was also suggested in the slide set.
 *
 * This exercise was executed on a MacBook Pro, with GeForce GT 650M
 * Using the CPU matrixMultiplication code, it takes about 18 seconds
 * Using this TILED approach, it only take about 0.13 ~0.15 seconds
 * 
 * See also:
 * Zhou Bin@ Nvidia & USTC, 2014, October, "CUDA Programming (2)" Lecture Slides
 * 
 *
 */

#include "stdio.h" 
#include "stdlib.h"

#include "hip/hip_runtime.h"



#define W 1024
#define TILE_WIDTH 16
#define DEBUG 1

void printMatrix(float *Matrix)
{
	const int MAX_SIZE_PRINTED = 4;
	
	printf("This is a %d by %d matrix.\n", W,W);
	
	if (W > MAX_SIZE_PRINTED) {
		printf("Actual displayed size is cut in 2 parts shown as");
		printf(" %d by %d matrix.\n", MAX_SIZE_PRINTED, MAX_SIZE_PRINTED);
		printf("   The Top_LEFT CORNER OF the %d * %d matrix:\n", W, W);
	}
	
	for(int i=0;i<W;i++)
	{
		  for(int j=0;j<W;j++)
		  	if(i < MAX_SIZE_PRINTED && j < MAX_SIZE_PRINTED){
		     if (DEBUG) printf("%5.2f ",*(Matrix+i*W+j));
			}
			if(i < MAX_SIZE_PRINTED && DEBUG) printf("\n");
	}
		
	if (W > MAX_SIZE_PRINTED){
		
		printf("   The LOWER_RIGHT CORNER OF the %d * %d matrix\n", W, W);
		
		for(int i=W-MAX_SIZE_PRINTED;i<W;i++)
		{
		  for(int j=W-MAX_SIZE_PRINTED;j<W;j++)
		  	if (DEBUG) printf("%5.2f ",*(Matrix+i*W+j));

			if(DEBUG) printf("\n");
		}
		
	}
		
}


/*
 *  This code is mostly copied from the slide set with some comments written by Ben Koo.
 *  
 *  In this test case, W = 1024, TILE_WIDTH = 16, making the dimGrid = 64 * 64
 *  Within each block, there are 16 * 16 threads.
 *
 *
 */
__global__ void matrixMulKernel_usingTile(float* Md, float* Nd, float* Pd, int Width)
{

	//This delcares the device memory as 16 * 16 float matrices
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // When W = 1024,t he block IDs (x * y) should be (64 * 64)
	int bx = blockIdx.x; int by = blockIdx.y;
	
    // When W = 1024, the thread IDs (x * y) should be (16 * 16)
	int tx = threadIdx.x; int ty = threadIdx.y;

	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;



	float PValue = 0;
	
	// When W = 1024, m should go from 0 to 63
	for (int m =0; m < Width/TILE_WIDTH; ++m){
	  // The following memory access takes place in shared memory
	  Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)];
	  Nds[ty][tx] = Nd[Col + (m*TILE_WIDTH+ty)*Width];
	  
	  //Make sure that all data are written in sync.
	  __syncthreads();

	  //Perform TILE level matrix multiplication and addition in synchrony.
	  for (int k = 0; k< TILE_WIDTH; ++k)	
		PValue += Mds[ty][k] * Nds[k][tx];
		__syncthreads();
	}
	
	//Take individually caldulated PValue and place it to the Pd (device memory array).
	Pd[Row * Width + Col] = PValue;
}
 
 
int main()
{
	int sNo = 0;
	hipSetDevice(sNo%8);

	int size = W*W*sizeof(float);
 
	float *M,*N,*P;
	float *d_M,*d_N,*d_P;

	M = (float *) malloc(size);
	N = (float *) malloc(size);
	P = (float *) malloc(size);

	hipMalloc((void **)&d_M,size);
	hipMalloc((void **)&d_N,size);
	hipMalloc((void **)&d_P,size);


    //Populate initial values to the M, N and P matrices
	for(int i=0;i<W*W;i++)
	{
	  *(M+i) = i;
      *(N+i) = i+1;
      *(P+i) = 0;
	}

    hipMemcpy(d_M, M,size,hipMemcpyHostToDevice);
    hipMemcpy(d_N, N,size,hipMemcpyHostToDevice);
	
	//Starting from here, set up CUDA timing mechanism
	float time_elapsed = 0;
	
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	dim3 dimGrid(W /TILE_WIDTH, W / TILE_WIDTH);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
		
	matrixMulKernel_usingTile<<< dimGrid, dimBlock >>>(d_M,d_N,d_P,W);
	
    hipEventRecord(stop,0);

	hipEventSynchronize(start);

	hipEventSynchronize(stop);

	//The following function returns time_elapsed using milli-seconds as time units
	hipEventElapsedTime(&time_elapsed, start, stop);
	
	//Finished timing for CUDA execution

    //To display time_elapsed into a number, divide it by 1000 first.
	printf("\n\nGPU Elapsed Time:%f\n", time_elapsed/1000);
	
        
    hipMemcpy(P,d_P,size,hipMemcpyDeviceToHost);

	printMatrix(P);
	
	free(M);free(N);free(P);
	hipFree(d_M);hipFree(d_N);hipFree(d_P);

	return 0;

}

