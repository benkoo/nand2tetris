#include "hip/hip_runtime.h"
#include "stdio.h" 
#include "stdlib.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#define W 8
#define TILE_WIDTH 2
#define DEBUG 1

void printMatrix(float *Matrix)
{
		for(int i=0;i<W;i++)
		{
		  for(int j=0;j<W;j++)
		     if (DEBUG) printf("%5.2f ",*(Matrix+i*W+j));
		  if (DEBUG) printf("\n");
		}
}

//This is for transposing a matrix
__global__ void transposeMatrix(float *oData, float *iData, int width, int height)
{
	int xIndex = blockIdx.x *TILE_WIDTH + threadIdx.x;
	int yIndex = blockIdx.y *TILE_WIDTH + threadIdx.y;

        int index_in  = xIndex + width  * yIndex;
	int index_out = yIndex + height * xIndex; 

	oData[index_out] = iData[index_in];

}

int matrixMul_cpu(float *M, float *N, float *P)
{
  for(int i=0;i<W;i++)
    for(int j=0;j<W;j++)
    {
	float sum=0;
	for (int k=0;k<W;k++)
	  {
	    float a = *(M+i*W+k);
        float b = *(N+k*W+j);
	    sum += a * b;
	  }
	*(P+i*W+j) = sum;

    }	

    return 0;  
}

__global__ void MatrixMulKernel_basic(float* Md, float* Nd, float* Pd, int Width)
{
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  
  float Pvalue = 0;
  if(Row < Width && Col < Width){
  for (int k = 0; k < Width; ++k)
	Pvalue += Md[Row * Width + k] * Nd[k * Width + Col];
	
  Pd[Row * Width + Col] = Pvalue;
  }
}

//Matrix Multiplication Kernel
__global__ void matrixMulKernel_1(float* Md, float* Nd, float* Pd, int Width)
{
	//2D Thread ID

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//Pvalue stores the Pd value computed by the thread
	float Pvalue = 0;

	for (int k = 0; k < Width; k++)
	{
	    float Mdelement = Md[ty * W + k];
	    float Ndelement = Nd[k * W + tx];
	    Pvalue += Mdelement * Ndelement;
	}	

	//Write the matrix to device memory each thread writes one element
	Pd[ty*Width + tx] = Pvalue;
}

__global__ void matrixMul_gpu(float *M,float *N, float *P, int width)
{
	int i = threadIdx.y;
	int j = threadIdx.x;


	float sum =0;
    for (int k = 0;k<width;k++)
	  {
	     float a = *(M+i*width+k);
             float b = *(N+k*width+j);
             sum += a*b;
	  }  

	*(P+i*width+j) = sum;

}

__global__ void matrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{

	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;



	float PValue = 0;
	for (int m =0; m < Width/TILE_WIDTH; ++m){
	  Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)];
	  Nds[ty][tx] = Nd[Col + (m*TILE_WIDTH+ty)*Width];
	  __syncthreads();

	  for (int k = 0; k< TILE_WIDTH; ++k)	
		PValue += Mds[ty][k] * Nds[k][tx];
		__syncthreads();
	}
	Pd[Row * Width + Col] = PValue;
}
 
 
int main()
{
	int sNo = 0;
	hipSetDevice(sNo%8);

	int size = W*W*sizeof(float);
 
	float *M,*N,*P,*T;
	float *d_M,*d_N,*d_P;

	M = (float *) malloc(size);
	N = (float *) malloc(size);
	P = (float *) malloc(size);
	T = (float *) malloc(size);

	hipMalloc((void **)&d_M,size);
	hipMalloc((void **)&d_N,size);
	hipMalloc((void **)&d_P,size);


	for(int i=0;i<W*W;i++)
	{
	  *(M+i) = i;
      *(N+i) = i+1;
      *(P+i) = 0;
      *(T+i) = 0;

	  if (DEBUG) printf("%f, %f ", *(M+i), *(N+i));
	  
	}
	  if (DEBUG) printf("\n");
	
	clock_t startT, finishT;

	startT = clock();
	int err = matrixMul_cpu(M,N,P);
	finishT = clock();
        printf("CPU elapsed time:%f\n\n", (float)(finishT - startT)/CLOCKS_PER_SEC);
		
	printMatrix(P);


    hipMemcpy(d_M, M,size,hipMemcpyHostToDevice);
    hipMemcpy(d_N, N,size,hipMemcpyHostToDevice);
	
	//Starting from here, set up the timing for CUDA devices
	float time_elapsed = 0;
	
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	dim3 dimGrid(W /TILE_WIDTH, W / TILE_WIDTH);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
		
	matrixMulKernel<<< dimGrid, dimBlock >>>(d_M,d_N,d_P,W);
	
    hipEventRecord(stop,0);

	hipEventSynchronize(start);

	hipEventSynchronize(stop);

	hipEventElapsedTime(&time_elapsed, start, stop);
	
	//Finished timing for CUDA execution

	printf("GPU Elapsed Time:%f\n", time_elapsed);
	
        
    hipMemcpy(P,d_P,size,hipMemcpyDeviceToHost);

	printMatrix(P);
	
	free(M);free(N);free(P);
	hipFree(d_M);hipFree(d_N);hipFree(d_P);

	return 0;

}

