#include "stdio.h"
#include "stdlib.h"

#include "hip/hip_runtime.h"


#define W 4


int matrixMul_cpu(float *M, float *N, float *P)
{
  for(int i=0;i<W;i++)
    for(int j=0;j<W;j++)
    {
	float sum=0;
	for (int k=0;k<W;k++)
	  {
	    float a = *(M+i*W+k);
            float b = *(N+k*W+j);
	    sum += a * b;
	  }
	*(P+i*W+j) = sum;

    }	

    return 0;  
}

__global__ void matrixMul_gpu(float *M,float *N, float *P, int width)
{
	int i = threadIdx.y;
	int j = threadIdx.x;
	
	float sum =0;
        for (int k = 0;k<width;k++)
	  {
	     float a = *(M+i*width+k);
             float b = *(N+k*width+j);
             sum += a*b;
	  }  
	*(P+i*width+j) = sum;
}



int main()
{

	int sNo = 0;

	hipSetDevice(sNo%8);

	int size = W*W*sizeof(float);
 
	float *M,*N,*P;
	float *d_M,*d_N,*d_P;

	M = (float *) malloc(size);
	N = (float *) malloc(size);
	P = (float *) malloc(size);
	
	hipMalloc((void **)&d_M,size);
	hipMalloc((void **)&d_N,size);
	hipMalloc((void **)&d_P,size);
	

	for(int i=0;i<W*W;i++)
	{
	  *(M+i) = i;
          *(N+i) = i+1;
          *(P+i) = 0;
	}

	hipMemcpy(d_M,M,size,hipMemcpyHostToDevice);
	hipMemcpy(d_N,N,size,hipMemcpyHostToDevice);

	
	int err = matrixMul_cpu(M,N,P);

	dim3 threadPerBlock(W,W);

	matrixMul_gpu<<< 1, threadPerBlock  >>>(d_M,d_N,d_P,W);
        
        hipMemcpy(P,d_P,size,hipMemcpyDeviceToHost);


	for(int i=0;i<W;i++)
	{
	  for(int j=0;j<W;j++)
	     printf("%f ",*(P+i*W+j));
	  printf("\n");
	}

	free(M);free(N);free(P);
	hipFree(d_M);hipFree(d_N);hipFree(d_P);

	return 0;

}
