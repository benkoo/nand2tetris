#include "hip/hip_runtime.h"
/**
 * Matrix multiplication Exercise : P = M . N. using Block-based shared memory
 *
 * This program basically follows the tutorial in class.
 *
 * Given the 1024*1024 matrix test case, this program got the best performance
 * boost using TILE_WIDTH = 16. This was also suggested in the slide set.
 *
 * This exercise was executed on a MacBook Pro, with GeForce GT 650M
 * Using the CPU matrixMultiplication code, it takes about 18 seconds
 * Using this Block-based approach, it only take about 0.13 ~0.15 seconds
 * 
 * See also:
 * Zhou Bin@ Nvidia & USTC, 2014, October, "CUDA Programming (2)" Lecture Slides
 * 
 *
 */

#include "stdio.h" 
#include "stdlib.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"


#define W 1024
#define TILE_WIDTH 16
#define DEBUG 1

void printMatrix(float *Matrix)
{
	const int MAX_SIZE_PRINTED = 4;
	
	printf("This is a %d by %d matrix.\n", W,W);
	
	if (W > MAX_SIZE_PRINTED) {
		printf("Actual displayed size is cut in 2 parts shown as");
		printf(" %d by %d matrix.\n", MAX_SIZE_PRINTED, MAX_SIZE_PRINTED);
		printf("   The Top_LEFT CORNER OF the %d * %d matrix:\n", W, W);
	}
	
	for(int i=0;i<W;i++)
	{
		  for(int j=0;j<W;j++)
		  	if(i < MAX_SIZE_PRINTED && j < MAX_SIZE_PRINTED){
		     if (DEBUG) printf("%5.2f ",*(Matrix+i*W+j));
			}
			if(i < MAX_SIZE_PRINTED && DEBUG) printf("\n");
	}
		
	if (W > MAX_SIZE_PRINTED){
		
		printf("   The LOWER_RIGHT CORNER OF the %d * %d matrix\n", W, W);
		
		for(int i=W-MAX_SIZE_PRINTED;i<W;i++)
		{
		  for(int j=W-MAX_SIZE_PRINTED;j<W;j++)
		  	if (DEBUG) printf("%5.2f ",*(Matrix+i*W+j));

			if(DEBUG) printf("\n");
		}
		
	}
		
}


/*
 *  This code is mostly copied from the slide set with some comments written by Ben Koo.
 *  
 *  In this test case, W = 1024, TILE_WIDTH = 16, making the dimGrid = 64 * 64
 *  Within each block, there are 16 * 16 threads.
 *
 *
 */
 
 //Matrix Multiplication Kernel
__global__ void matrixMulKernel_block(float* Md, float* Nd, float* Pd, int Width)
{
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  
  float Pvalue = 0;
  
  //Only calculate values when Row and Col are smaller than Width
  //Otherwise there might be some threads that are beyond the bounds of 
  //the desirable matrix size.
  if(Row < Width && Col < Width){
  for (int k = 0; k < Width; ++k)
	Pvalue += Md[Row * Width + k] * Nd[k * Width + Col];
	
  Pd[Row * Width + Col] = Pvalue;
  }
}
 
//Matrix Multiplication Kernel
__global__ void matrixMulKernel_1(float* Md, float* Nd, float* Pd, int Width)
{

	__shared__ float Ms[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float Ns[BLOCK_SIZE][BLOCK_SIZE];
			
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;
	
	//2D Thread ID

	int tx = threadIdx.x;
	int ty = threadIdx.y;


    const int BLOCK_SIZE = 16;
	int wA = TILE_WIDTH;
	int wB = TILE_WIDTH;
	
	// Index of the first sub-matrix of M processed by the block
	int aBegin = wA * BLOCK_SIZE * by;

	// Index of the last sub-matrix of M processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of M
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of N processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of N
    int bStep  = BLOCK_SIZE * wB;
		
	//Pvalue stores the Pd value computed by the thread
	float Pvalue = 0;

    // Loop over all the sub-matrices of M and N
    // required to compute the block sub-matrix

    for (int a = aBegin, b = bBegin;
	         a <= aEnd;
	         a += aStep, b += bStep)
    {

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        Ms[ty][tx] = Md[a + wA * ty + tx];
        Ns[ty][tx] = Nd[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
	       Pvalue += Ms[ty][k] * Ns[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
	}

	   // Write the block sub-matrix to device memory;
	   // each thread writes one element
	    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	    Pd[c + wB * ty + tx] = Pvalue;
}


__global__ void matrixMulKernel_usingTile(float* Md, float* Nd, float* Pd, int Width)
{

	//This delcares the device memory as 16 * 16 float matrices
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // When W = 1024, the block IDs (x * y) should be (64 * 64)
	int bx = blockIdx.x; int by = blockIdx.y;
	
    // When W = 1024, the thread IDs (x * y) should be (16 * 16)
	int tx = threadIdx.x; int ty = threadIdx.y;

	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;



	float PValue = 0;
	
	// When W = 1024, m should go from 0 to 63
	for (int m =0; m < Width/TILE_WIDTH; ++m){
	  // The following memory access takes place in shared memory
	  Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)];
	  Nds[ty][tx] = Nd[Col + (m*TILE_WIDTH+ty)*Width];
	  
	  //Make sure that all data are written in sync.
	  __syncthreads();

	  //Perform TILE level matrix multiplication and addition in synchrony.
	  for (int k = 0; k< TILE_WIDTH; ++k)	
		PValue += Mds[ty][k] * Nds[k][tx];
		__syncthreads();
	}
	
	//Take individually caldulated PValue and place it to the Pd (device memory array).
	Pd[Row * Width + Col] = PValue;
}
 
 
int main()
{
	int sNo = 0;
	hipSetDevice(sNo%8);

	int size = W*W*sizeof(float);
 
	float *M,*N,*P;
	float *d_M,*d_N,*d_P;

	M = (float *) malloc(size);
	N = (float *) malloc(size);
	P = (float *) malloc(size);

	hipMalloc((void **)&d_M,size);
	hipMalloc((void **)&d_N,size);
	hipMalloc((void **)&d_P,size);


    //Populate initial values to the M, N and P matrices
	for(int i=0;i<W*W;i++)
	{
	  *(M+i) = i;
      *(N+i) = i+1;
      *(P+i) = 0;
	}

    hipMemcpy(d_M, M,size,hipMemcpyHostToDevice);
    hipMemcpy(d_N, N,size,hipMemcpyHostToDevice);
	
	//Starting from here, set up CUDA timing mechanism
	float time_elapsed = 0;
	
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	dim3 dimGrid(W /TILE_WIDTH, W / TILE_WIDTH);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
		
	matrixMulKernel_1<<< dimGrid, dimBlock >>>(d_M,d_N,d_P,W);

		
    hipEventRecord(stop,0);

	hipEventSynchronize(start);

	hipEventSynchronize(stop);

	//The following function returns time_elapsed using milli-seconds as time units
	hipEventElapsedTime(&time_elapsed, start, stop);
	
	//Finished timing for CUDA execution

    //To display time_elapsed into a number, divide it by 1000 first.
	printf("\n\nGPU Elapsed Time:%f\n", time_elapsed/1000);
	
        
    hipMemcpy(P,d_P,size,hipMemcpyDeviceToHost);

	printMatrix(P);
	
	free(M);free(N);free(P);
	hipFree(d_M);hipFree(d_N);hipFree(d_P);

	return 0;

}

